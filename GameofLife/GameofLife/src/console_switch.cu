﻿#include "console_switch.cuh"

#include "system_state.cuh"
#include "world_generation.cuh"

void ConsoleSwitch::printMenu()
{
    std::cout << "Enter grid size: ";
    std::cin >> grid_width;
    std::cout << "Available modes of Operation: " << std::endl;
    std::cout << "1. CPU single thread" << std::endl;
    std::cout << "2. CPU multi thread" << std::endl;
    std::cout << "3. GPU CUDA" << std::endl;
    std::cout << "Enter mode of operation: ";
    int mode_input;
    std::cin >> mode_input;
    switch(mode_input)
    {
    case 1:
        system_mode = CPU_SINGLE_THREAD;
        std::cout << "CPU single thread mode selected" << std::endl;
        break;
    case 2:
        system_mode = CPU_MULTI_THREAD;
        std::cout << "CPU multi thread mode selected" << std::endl;
        break;
    case 3:
        system_mode = GPU_CUDA;
        std::cout << "GPU CUDA mode selected" << std::endl;
        break;
    default:
        system_mode = CPU_SINGLE_THREAD;
        std::cout << "Invalid mode selected, defaulting to CPU single thread mode" << std::endl;
        break;
    }
    std::cout << "World grid is randomly generated" << std::endl;
    WorldGeneration::generate_random_world();
}

