﻿#include "cpu_compute.cuh"

#include <thread>

#include "system_state.cuh"

unsigned int processor_count;

void ThreadWork(WorldGrid* world_grid, WorldGrid* world_buffer, int start)
{
    for(int i = start; i < start + world_grid->getWorldSize() / processor_count; i++)
    {
        const int x_cord = i % world_grid->getGridWidth();
        const int y_cord = i / world_grid->getGridWidth();
			
        int alive_neighbours = 0; 
        for(int j = -1; j < 2; j++)
        {
            for(int k = -1; k < 2; k++)
            {
                if(j == 0 && k == 0)
                {
                    continue;
                }
                if(world_grid->getCell(x_cord + j, y_cord + k) == ALIVE)
                {
                    alive_neighbours++;
                }
            } 
        }
        if(alive_neighbours == 3) world_buffer->setCell(x_cord, y_cord, ALIVE);
        else if(alive_neighbours == 2) world_buffer->setCell(x_cord, y_cord, world_grid->getCell(x_cord, y_cord));
        else world_buffer->setCell(x_cord, y_cord, DEAD);
    }
}

ThreadManager::ThreadManager(WorldGrid* world_grid)
{
    cpu_state = INIT;
    init(world_grid);
}

void ThreadManager::init(WorldGrid* _world_grid)
{
    cpu_state = RUNNIG;
    switch(system_mode)
    {
    case CPU_SINGLE_THREAD:
        processor_count = 1;
        break;
    case CPU_MULTI_THREAD:
        processor_count = std::thread::hardware_concurrency();
        break;
    }
    world_grid = _world_grid;
    world_buffer = new WorldGrid(world_grid->getGridWidth());
}

void ThreadManager::run()
{
    while(cpu_state == RUNNIG)
    {
        const auto threads = new std::thread[processor_count];
			
        // Initializing threads
        //int starting_point = i*(world_grid->getWorldSize()/processor_count)); <- This is the starting point for each thread. This was used in the previous version of the code. Now inlined
        for(int i = 0; i < processor_count; i++) threads[i] = std::thread(ThreadWork, world_grid, world_buffer, i*(world_grid->getWorldSize()/processor_count));
			
        // Joining threads
        for(int i = 0; i < processor_count; i++) threads[i].join();
        world_grid->printWorld();
        // Swap the buffers and threads clean up
        delete[] threads;
        delete world_grid;
        world_grid = world_buffer;
        world_buffer = new WorldGrid(world_grid->getGridWidth());
    }
}

void ThreadManager::cleanup()
{
    cpu_state = CLOSING;
    delete world_grid;
    delete world_buffer;
}
