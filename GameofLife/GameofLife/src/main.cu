﻿#include <iostream>

#include "app.cuh"
#include "system_state.cuh"
#include "world_generation.cuh"
#include "world_grid.cuh"

int main(int argc, char **argv) {
    grid_width = 10;
    
    // Initialize the app
    App::init();

    // Run the app
    App::run();

    // Clean up
    App::cleanup();

    // WorldGeneration::generate_random_world();
    // grid -> printWorld();
    // std::cin.get();
    

    return 0;
}
