#include "hip/hip_runtime.h"
﻿#include "gpu_compute.cuh"
#include <cstdlib>
#include <iostream>
#include <string>
#include <thread>
#include "hip/hip_runtime.h"
#include ""
#include "system_state.cuh"


__global__ void compute(bool *primary_device_buffer, bool *secondary_device_buffer, int world_width)
{
    int index = blockIdx.x * blockDim.x +threadIdx.x;
    const int x_cord = index % world_width;
    const int y_cord = index / world_width;
    int alive_neighbours = 0;

    for (int k = -1; k < 2; k++)
    {
        for (int j = -1; j < 2; j++)
        {
            if (j == 0 && k == 0)
            {
                continue;
            }
            if (primary_device_buffer[(y_cord + k) * world_width + (x_cord + j)] == true)
            {
                alive_neighbours++;
            }
        }
    }
    if(alive_neighbours < 2 || alive_neighbours > 3)
    {
        secondary_device_buffer[index] = false;
    }
    else if(alive_neighbours == 3)
    {
        secondary_device_buffer[index] = true;
    }
    else
    {
        secondary_device_buffer[index] = primary_device_buffer[index];
    }
}

GpuCompute::GpuCompute(int world_width, bool* start_world_frame)
{
    gpu_state = INIT;
    this->world_width = world_width;
    this->host_world_buffer = start_world_frame;
}

void GpuCompute::init()
{
    buffer_size = world_width * world_width * sizeof(bool);
    //GPU device memory allocation
    hipMalloc(&primary_device_world_buffer, buffer_size);
    hipMalloc(&secondary_device_world_buffer, buffer_size);

    //Copy world buffer to GPU device
    hipMemcpy(primary_device_world_buffer, host_world_buffer, buffer_size,hipMemcpyHostToDevice);

    gpu_state = RUNNIG;
    
}

void print(int width, bool* buffer)
{
    std::string frame;
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            std::string cell = buffer[i * width + j] ? "X" : " ";
            frame.append(cell);
            frame.append(" ");
        }
        frame.append("\n");
    }
    std::cout << frame;
    std::this_thread::sleep_for(std::chrono::milliseconds(20));
    system("CLS"); 
    
}

void GpuCompute::run()
{
    while (gpu_state == RUNNIG)
    {
        //print(world_width, host_world_buffer);
        compute <<< world_width, world_width >>> (primary_device_world_buffer, secondary_device_world_buffer, world_width);
        hipMemcpy(host_world_buffer, secondary_device_world_buffer, buffer_size, hipMemcpyDeviceToHost);
        hipMemcpy(primary_device_world_buffer, secondary_device_world_buffer, buffer_size, hipMemcpyDeviceToDevice);
        print(world_width, host_world_buffer);
    }
}

void GpuCompute::cleanup()
{
    free(host_world_buffer);
    hipFree(primary_device_world_buffer);
    hipFree(secondary_device_world_buffer);
    gpu_state = CLOSING;
}


