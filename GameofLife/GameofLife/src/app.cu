﻿#include "app.cuh"

#include "console_switch.cuh"
#include "cpu_compute.cuh"
#include "gpu_compute.cuh"
#include "system_state.cuh"

ThreadManager *thread_manager;

int App::init()
{
    system_state = INIT;
    ConsoleSwitch::printMenu();
    return 0;
}

int App::run()
{
    switch (system_mode)
    {
    case CPU_SINGLE_THREAD:
        {
            thread_manager = new ThreadManager(grid);
            thread_manager->run();
            break;
            
        }
    case CPU_MULTI_THREAD:
        {
            thread_manager = new ThreadManager(grid);
            thread_manager->run();
            break;
        }
    case GPU_CUDA:
        {
            auto *compute_gpu = new GpuCompute(grid_width,grid->getGrid());
            compute_gpu->init();
            compute_gpu->run();
            break;
            
        }
    }
    return 0;
}

int App::cleanup()
{
    return 0;
}

