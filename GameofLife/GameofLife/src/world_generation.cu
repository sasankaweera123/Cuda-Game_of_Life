﻿#include "world_generation.cuh"

#include <cstdlib>

#include "system_state.cuh"
#include "world_grid.cuh"

void WorldGeneration::generate_random_world()
{
    grid = new WorldGrid(grid_width);
    for(int i = 0; i < grid_width; i++)
    {
        for(int j = 0; j < grid_width; j++)
        {
            grid->setCell(i, j, static_cast<bool>(rand() % 2));
        }
    }
    
}
