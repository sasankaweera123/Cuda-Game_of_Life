﻿#include "world_grid.cuh"
#include <iostream>
#include <thread>

// world grid constructor
WorldGrid::WorldGrid(int width)
{
    grid_width = width;
    grid = new bool[width * width];
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            setCell(i, j, false);
        }
    }
}

void WorldGrid::setCell(int x, int y, bool value)
{
    grid[x * grid_width + y] = value;
    
}
// world grid destructor
WorldGrid::~WorldGrid()
{
    delete[] grid;
}

int WorldGrid::getGridWidth() const
{
    return grid_width;
}

bool* WorldGrid::getGrid() const
{
    return grid;
}

bool WorldGrid::getCell(int x, int y) const
{
    try
    {
        return grid[x * grid_width + y];
    }catch (...)
    {
        return false;
    }
}

int WorldGrid::getWorldSize() const
{
    return grid_width* grid_width;
}

void WorldGrid::printWorld() const
{
    std::string frame;
    for(int i = 0; i < getGridWidth(); i++)
    {
        for (int j=0; j < getGridWidth(); j++)
        {
            std::string cell = getCell(i, j) ? "X" : "0";
            frame.append(cell);
            frame.append(" ");
        }
        frame.append("\n");
    }
    std::cout << frame << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(50));
    //system("CLS");
}